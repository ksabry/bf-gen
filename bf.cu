#include "hip/hip_runtime.h"
// High level notes
//   Frankly speaking, simulation of a language is not the best suited to gpu execution and comes with some serious considerations:
//     The completely generic control flow is fundamentally a loop in which each iteration is dependent on the last
//     This alone precludes many optimizations which are typically desirable; with how generic things are the loop cannot be parallelized (save _maybe_ certain explicitly defined special cases)
//     Since we do want to have a maximum execution time per program, the loop length does indeed have an upper bound, which, if structured appropriately, could afford some optimization in the way of loop unrolling
//     However, on the surface of it this may not be compatible with conditional short-circuiting of the loop which itself is desirable enough to probably not be worth giving up (many programs will end very quickly)
//     That said, highly controlled partial loop unrollment (essentially implemented as a nested loop which runs a constant number of times) is also an option which is worth exploring,
//     the inner loop would essentially no-op on each loop execution after it has short-circuted.
//     I suppose it is possible that cuda would try something like this anyway, and while I find it unlikely that it is smart enough to figure it out in this case it should be investigated before attempting alternatives
//
//     An interesting (crazy) thought is to attempt to directly do speculative execution and/or branch prediction with the bf program's control flow.
//     This may not be as intractable as it first seems due to the fact that the control flow is both quite simple and represented directly (all cases are a zero/non-zero jump),
//     as well as the fact that the theoretical pipelining _should_ be somewhat straightforward
//
//     Another thought is to actually directly compile the bf programs to gpu assembly before execution. My intuition is that this per-program compilation overhead would quickly overwhelm the increased "runtime" optimization
//     if this were to be done the compilation would have to be extremely simple, probably close to 1-to-1
//     That said, bf is simple enough that a somewhat 1-to-1 compilation would theoretically be done quite quickly, so depending on how fast the gpu assembly can emit this _maybe_ could work
//     Another huge confounder is that this would require seperate executable memory to be uploaded per thread/block, I have no idea if that is even possible without multiple host calls
//     I suppose to the last point, this could also be done with regular cpu assembly or an intermediate like llvm, which might be an interesting avenue
//
//   Having a good profiling baseline will be very important if we actually get serious about attempting to optimize this down


// Input format includes the following

// chunk_count_per_program:
//   represents the number of program chunks per program
//   1d array of uint8, dimension [PROGRAM_COUNT_PER_BLOCK]

// data_ops:
//   represents a sequence of +-<> ops
//   3d array of uint8, dimension [DATA_CELL_COUNT+1][MAX_CHUNK_COUNT_PER_PROGRAM][PROGRAM_COUNT_PER_BLOCK]
//     each program member array at index [program_index] of dimension [DATA_CELL_COUNT+1][MAX_CHUNK_COUNT_PER_PROGRAM] only has a meaningful size of chunk_count_per_program[program_index]
//     each program chunk member array at index [program_index][program_chunk_index] of dimension [DATA_CELL_COUNT+1] has the following format:
//       [ data_ptr_diff, data_cell_0_diff, data_cell_1_diff, ..., data_cell_{DATA_CELL_COUNT-1}_diff ]
//       the first element is data_ptr_diff, this could be moved into its own array, however my thought is that locality of data is relevent here, especially since modification is a simple add similar to data modification
//       that said there may not be caching concerns as this should probably be only relevent in a single thread, and the volume of data per program is relatively small, worth testing a version which separates this

// io_ops:
//   represents a . or , op
//   2d array of IO_OP_t, dimension [MAX_CHUNK_COUNT_PER_PROGRAM][PROGRAM_COUNT_PER_BLOCK]
//     each program member array at index [program_index] of dimension [MAX_CHUNK_COUNT_PER_PROGRAM] only has a meaningful size of chunk_count_per_program[program_index]
//     the members of this array each associated with a single program chunk represent a read from input (,) to the current data cell or a write to output (.) from the current data cell
//       in both cases, the respective io pointer will be incremented and wrap

// control_ops:
//   represents a [ or ] op
//   3d array of uint8, dimension [2][MAX_CHUNK_COUNT_PER_PROGRAM][PROGRAM_COUNT_PER_BLOCK]
//     each program member array at index [program_index] of dimension [2][MAX_CHUNK_COUNT_PER_PROGRAM] only has a meaningful size of chunk_count_per_program[program_index]
//     each program chunk member array at index [program_index][program_chunk_index] of dimension [2] represents the next chunk if the current data cell is 0, and the next chunk if the current data cell is nonzero
//       these members are chunk indexes within the current program and are guaranteed to be less than or equal to chunk_count_per_program[program_index]
//       if the next index is equla to chunk_count_per_program[program_index] the program terminates

// NOTE: the following could be made significantly more memory efficient if we wish to have the same input data for each block, or even for the entire block grid, though there may be data access speed implications, especially in the latter case

// input_data_count_per_program
//   represents the length of input_data per program
//   1d array of uint8, dimension [PROGRAM_COUNT_PER_BLOCK]

// input_data
//   represents the sequence to be read using ,
//   2d array of uint8, dimension [MAX_INPUT_DATA_COUNT][PROGRAM_COUNT_PER_BLOCK]
//     each program member array at index [program_index] of dimension [MAX_INPUT_DATA_COUNT] is 0 terminated and only has a meaningful size up to the first 0
//     the input_data_ptr will wrap to the beginning whenever it encounters a 0


// Execution notes

// The execution flow is as follows:
//   data_ops[current_chunk_index] is applied to the working data_cell_array
//   io_ops[current_chunk_index] is applied if present, copying between the current data cell, input stream, or output stream as necessary
//   control_ops[current_chunk_index] is used to find the next chunk to execute, if the next chunk is out of bounds or if the chunks_executed_counter reaches MAX_CHUNKS_EXECUTED_COUNTER the program is terminated

// it might be worth experimenting with uint16 or uint32 rather than uint8 for potential speedups regarding data alignment,
//   I do not know how important data alignment for integral operations is on the gpu as opposed to the cpu, however my intuition is that the gpu should support per byte operations decently well, and that the memory overhead would not justify these changes
//   in regards to this remember that memory overhead can translate to performance overhead very easily when factoring in per thread or per block processor caches as well as actual coprocessor data transfer

#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

// NOTE: We use preprocessing macros purely for performance reasons

#define PROGRAM_COUNT_PER_BLOCK 256
#define DATA_CELL_COUNT 31 /* NOTE: this plus 1 should be 4 byte aligned; might be particularly worth attempting 15, as cuda can handle up to 16 bytes in a single instruction */
#define MAX_CHUNK_COUNT_PER_PROGRAM 16 /* NOTE: we can increase this if necessary, but every programs' memory scales linearly with this no matter how much of it they use */
#define MAX_INPUT_DATA_COUNT 16
#define MAX_OUTPUT_DATA_COUNT 16
#define MAX_CHUNKS_EXECUTED_COUNTER 1024 /* NOTE: we can experiment a good deal with this value */

// TODO: enum is probably the same as this after compile and definitely better practice
#define IO_OP_t uint8_t
#define IO_OP_NONE 0
#define IO_OP_INPUT 1
#define IO_OP_OUTPUT 2

#define WORKING_DATA_PTR_OFFSET DATA_CELL_COUNT
#define DATA_CELL_MIN (DATA_CELL_COUNT / 2)
#define DATA_CELL_MAX (3 * DATA_CELL_COUNT / 2)
#define DATA_OP_OFFEST (DATA_CELL_COUNT / 2)

// TODO: current version has a lot of problems as discussed in various comments and the wall of text above, however I want to get a simple running version before prematurely addressing them
__global__ void ExecuteBfKernal(
	uint8_t chunk_count_per_program[PROGRAM_COUNT_PER_BLOCK],
	uint8_t data_ops[DATA_CELL_COUNT+1][MAX_CHUNK_COUNT_PER_PROGRAM][PROGRAM_COUNT_PER_BLOCK],
	IO_OP_t io_ops[MAX_CHUNK_COUNT_PER_PROGRAM][PROGRAM_COUNT_PER_BLOCK],
	uint8_t control_ops[2][MAX_CHUNK_COUNT_PER_PROGRAM][PROGRAM_COUNT_PER_BLOCK],
	uint8_t input_data_count_per_program[PROGRAM_COUNT_PER_BLOCK],
	uint8_t input_data[MAX_INPUT_DATA_COUNT][PROGRAM_COUNT_PER_BLOCK],
	uint8_t output_data[MAX_OUTPUT_DATA_COUNT][PROGRAM_COUNT_PER_BLOCK],
)
{
	uint8_t program_index = threadIdx.x;
	uint8_t current_chunk_ptr = 0;

	uint8_t current_working_data_ptr = WORKING_DATA_PTR_OFFSET;
	// Figure out if this needs to be explicitly zero initialized
	uint8_t current_working_data[DATA_CELL_COUNT * 2];
	
	uint8_t current_input_data_ptr = 0;
	uint8_t current_output_data_ptr = 0;

	// NOTE: see high level notes for my thoughts on the potential performance optimizations of this loop
	for (uint32_t i = MAX_CHUNKS_EXECUTED_COUNTER; i > 0; i--)
	{
		// execute data ops
		// NOTE: this might be worth explicitely splitting up over a warp, or some form of cooperative group, though this is certainly non-trivial
		for (uint32_t working_data_index = 1; i < DATA_CELL_COUNT; i++)
		{
			current_working_data[current_working_data_ptr + working_data_index - DATA_OP_OFFEST - 1] += data_ops[program_index][current_chunk_ptr][working_data_index];
		}
		current_working_data_ptr += data_ops[program_index][current_chunk_ptr][0];

		// verify data pointer is still in bounds
		// NOTE: this worries me perf-wise
		if (current_working_data_ptr < DATA_CELL_MIN || current_working_data_ptr >= DATA_CELL_MAX)
		{
			break;
		}

		// execute io ops
		// NOTE: cuda should be smart enough not to emit diverging branches here, but this should probably be confirmed
		IO_OP_t io_op = io_ops[program_index][current_chunk_ptr];
		if (io_op == IO_OP_INPUT)
		{
			current_working_data[current_working_data_ptr] = input_data[program_index][current_input_data_ptr];
			current_input_data_ptr = (current_input_data_ptr + 1) % input_data_count_per_program[program_index];
		}
		if (io_op == IO_OP_OUTPUT)
		{
			current_output_data[current_output_data_ptr] = current_working_data[current_working_data_ptr];
			current_output_data_ptr = (current_output_data_ptr + 1) % MAX_OUTPUT_DATA_COUNT;
		}

		// execute control flow ops
		// NOTE: even if cuda is smart enough not to emit diverging branches here, I think it's better to just access using a conditional directly
		current_chunk_ptr = control_ops[program_index][program_chunk_index][ current_working_data[current_working_data_ptr] != 0 ]
		// Check if program has regularly termintated
		// NOTE: this worries me perf-wise
		if (current_chunk_ptr >= chunk_count_per_program[program_index])
		{
			break;
		}
	}
}

struct ExecuteBfParams
{
	uint8_t chunk_count_per_program[PROGRAM_COUNT_PER_BLOCK],
	uint8_t data_ops[DATA_CELL_COUNT+1][MAX_CHUNK_COUNT_PER_PROGRAM][PROGRAM_COUNT_PER_BLOCK],
	IO_OP_t io_ops[MAX_CHUNK_COUNT_PER_PROGRAM][PROGRAM_COUNT_PER_BLOCK],
	uint8_t control_ops[2][MAX_CHUNK_COUNT_PER_PROGRAM][PROGRAM_COUNT_PER_BLOCK],
	uint8_t input_data_count_per_program[PROGRAM_COUNT_PER_BLOCK],
	uint8_t input_data[MAX_INPUT_DATA_COUNT][PROGRAM_COUNT_PER_BLOCK],
	uint8_t output_data[MAX_OUTPUT_DATA_COUNT][PROGRAM_COUNT_PER_BLOCK],
}

void ExecuteBfCuda(ExecuteBfCudaParams & params)
{
	// Essentially just moves data from host to device and back after execution

	uint8_t* DEVICE_chunk_count_per_program;
	uint8_t* DEVICE_data_ops;
	IO_OP_t* DEVICE_io_ops;
	uint8_t* DEVICE_control_ops;
	uint8_t* DEVICE_input_data_count_per_program;
	uint8_t* DEVICE_input_data;
	uint8_t* DEVICE_output_data;

	hipMalloc(&DEVICE_chunk_count_per_program, PROGRAM_COUNT_PER_BLOCK);
	hipMalloc(&DEVICE_data_ops, (DATA_CELL_COUNT+1) * MAX_CHUNK_COUNT_PER_PROGRAM * PROGRAM_COUNT_PER_BLOCK);
	hipMalloc(&DEVICE_io_ops, MAX_CHUNK_COUNT_PER_PROGRAM * PROGRAM_COUNT_PER_BLOCK);
	hipMalloc(&DEVICE_control_ops, 2 * MAX_CHUNK_COUNT_PER_PROGRAM * PROGRAM_COUNT_PER_BLOCK);
	hipMalloc(&DEVICE_input_data_count_per_program, PROGRAM_COUNT_PER_BLOCK);
	hipMalloc(&DEVICE_input_data, MAX_INPUT_DATA_COUNT * PROGRAM_COUNT_PER_BLOCK);
	hipMalloc(&DEVICE_output_data, MAX_OUTPUT_DATA_COUNT * PROGRAM_COUNT_PER_BLOCK);

	hipMemcpy(DEVICE_chunk_count_per_program, params.chunk_count_per_program, PROGRAM_COUNT_PER_BLOCK, hipMemcpyHostToDevice);
	hipMemcpy(DEVICE_data_ops, params.data_ops, (DATA_CELL_COUNT+1) * MAX_CHUNK_COUNT_PER_PROGRAM * PROGRAM_COUNT_PER_BLOCK, hipMemcpyHostToDevice);
	hipMemcpy(DEVICE_io_ops, params.io_ops, MAX_CHUNK_COUNT_PER_PROGRAM * PROGRAM_COUNT_PER_BLOCK, hipMemcpyHostToDevice);
	hipMemcpy(DEVICE_control_ops, params.control_ops, 2 * MAX_CHUNK_COUNT_PER_PROGRAM * PROGRAM_COUNT_PER_BLOCK, hipMemcpyHostToDevice);
	hipMemcpy(DEVICE_input_data_count_per_program, params.input_data_count_per_program, PROGRAM_COUNT_PER_BLOCK, hipMemcpyHostToDevice);
	hipMemcpy(DEVICE_input_data, params.input_data, MAX_INPUT_DATA_COUNT * PROGRAM_COUNT_PER_BLOCK, hipMemcpyHostToDevice);

	hipMemset(DEVICE_output_data, 0, MAX_OUTPUT_DATA_COUNT * PROGRAM_COUNT_PER_BLOCK);

	ExecuteBf<<<1, PROGRAM_COUNT_PER_BLOCK>>>(
		DEVICE_chunk_count_per_program,
		DEVICE_data_ops,
		DEVICE_io_ops,
		DEVICE_control_ops,
		DEVICE_input_data_count_per_program,
		DEVICE_input_data,
		DEVICE_output_data,
	);

	hipMemcpy(params.output_data, DEVICE_output_data, MAX_OUTPUT_DATA_COUNT * PROGRAM_COUNT_PER_BLOCK, hipMemcpyDeviceToHost);

	hipFree(DEVICE_chunk_count_per_program);
	hipFree(DEVICE_data_ops);
	hipFree(DEVICE_io_ops);
	hipFree(DEVICE_control_ops);
	hipFree(DEVICE_input_data_count_per_program);
	hipFree(DEVICE_input_data);
	hipFree(DEVICE_output_data);
}

void BfSourceToExecuteParams(char const * const source, ExecuteBfParams& params, uint32_t program_index)
{
	char source_char;
	uint32_t source_index = 0;
	
	uint8_t current_chunk_index = 0;
	uint8_t current_chunk_data_op[DATA_CELL_COUNT];
	memset(current_chunk_data_op, 0, DATA_CELL_COUNT);
	uint8_t current_chunk_data_ptr = DATA_OP_OFFEST;

	while (source_char = source[source_index++])
	{
		switch (source_char)
		{
			case '+':
				current_chunk_data_op[current_chunk_data_ptr] += 1;
				break;

			case '-':
				current_chunk_data_op[current_chunk_data_ptr] -= 1;
				break;

			case '>':
				current_chunk_data_ptr += 1;
				if (current_chunk_data_ptr == DATA_CELL_COUNT - 1) 
				{
					// TODO: fail gracefully
					throw 1;
				}
				break;

			case '<':
				if (current_chunk_data_ptr == 0)
				{
					// TODO: fail gracefully
					throw 1;
				}
				current_chunk_data_ptr -= 1;
				break;

			case '.':
				// commit data op
				uint8_t* param_dest = &(params.data_ops[program_index][current_chunk_index][1])
				memcpy(current_chunk_data_op, param_dest, DATA_CELL_COUNT);
				params.data_ops[program_index][current_chunk_index][0] = current_chunk_data_ptr;

				// clear data op
				memset(current_chunk_data_op, 0, DATA_CELL_COUNT);
				current_chunk_data_ptr = DATA_OP_OFFEST;

				
		}
	}
}

// This should not be used for actual execution as it loads only a single program onto the gpu, it's only purpose is a quick test functionality
void ExecuteBfSingle(char const * const program)
{

}

int main()
{

}
